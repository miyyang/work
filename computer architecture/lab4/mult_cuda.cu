// matrix multiplication
// CA_LAB4

#include<stdio.h>
#include<iostream>
#include<cstdlib>
#include<time.h>
#include<hip/hip_runtime.h>

#define TILE_WIDTH 32
#define DEBUG 0
using namespace std;

void print(float *A, int n, int m)
{
	for (int i = 0; i < n; i++)
		for (int j = 0; j < m; j++)
		cout << A[n*i+j] << " ";
		cout<<endl;
}

void init_matrix (float *mat, float value, int n, int m)
{
	int size = n * m; 
	for (int i = 0; i < size; i++)
		mat[i] = value;
}

void multMatrixSeq (float *mA, float *mB, float *mC, int n, int m, int o)
{
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < o; j++)
		{
			float sum = 0;
			for (int k = 0; k < m; k++)
			{
				sum += mA[m*i+k] * mB[o*k+j];
			}
			mC[o*i+j] = sum;
		}
	}
}

__global__ void CU_multMatrixThread (float *mA, float *mB, float *mC, int n, int m, int o)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if ((row<n) && (col<o))
	{
		float temp = 0;
		for (int i = 0; i < m; i++)
			temp += mA[row*m+i] * mB[i*o+col];
		mC[row*o+col] = temp;
	}
}

__global__ void CU_multMatrixTiled(float *mA, float *mB, float *mC, int n, int m, int o){
	__shared__ float tmpM1[TILE_WIDTH][TILE_WIDTH];
	__shared__ float tmpM2[TILE_WIDTH][TILE_WIDTH];

	int bx = blockIdx.x;
	int by = blockIdx.y;
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * TILE_WIDTH + ty;
	int col = bx * TILE_WIDTH + tx;

	float Pvalue = 0;

	for (int k = 0; k < (m + TILE_WIDTH - 1) / TILE_WIDTH; ++k)
	{
		if (k*TILE_WIDTH + tx < m && row < n)
			tmpM1[ty][tx] = mA[row * m + k*TILE_WIDTH + tx];
		else
			tmpM1[ty][tx] = 0;

		if (k*TILE_WIDTH + ty < m && col < o)
			tmpM2[ty][tx] = mB[(k*TILE_WIDTH + ty) * o + col];
		else
			tmpM2[ty][tx] =0;

		__syncthreads();
		
		for(int k = 0; k < TILE_WIDTH; ++k)
			Pvalue += tmpM1[ty][k] * tmpM2[k][tx];
		
		__syncthreads();
	}

	if (row < n && col < o)
		mC[row * o + col] = Pvalue;
}

void multMatrixTiled(float *A, float *B, float *C, int n, int m, int o)
{
	float blockSize = TILE_WIDTH;
	float *mA, *mB, *mC;

	hipMalloc(&mA, n * m * sizeof(float));
	hipMalloc(&mB, m * o * sizeof(float));
	hipMalloc(&mC, n * o * sizeof(float));

	hipMemcpy(mA, A, n * m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mB, B, m * o * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(blockSize,blockSize,1);
	dim3 blocks(ceil(o/blockSize),ceil(n/blockSize),1);
	CU_multMatrixThread<<<blocks,threads>>>(mA,mB,mC,n,m,o);

	hipMemcpy (C, mC, n * o * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(mA);
	hipFree(mB);
	hipFree(mC);

}

void multMatrixThread(float *A, float *B, float *C, int n, int m, int o)
{
	float blockSize = TILE_WIDTH;
	float *mA, *mB, *mC;

	hipMalloc(&mA, n * m * sizeof(float));
	hipMalloc(&mB, m * o * sizeof(float));
	hipMalloc(&mC, n * o * sizeof(float));

	hipMemcpy(mA, A, n * m * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(mB, B, m * o * sizeof(float), hipMemcpyHostToDevice);

	dim3 threads(blockSize,blockSize,1);
	dim3 blocks(ceil(o/blockSize),ceil(n/blockSize),1);
	CU_multMatrixThread<<<blocks,threads>>>(mA,mB,mC,n,m,o);

	hipMemcpy (C, mC, n * o * sizeof(float), hipMemcpyDeviceToHost);

	hipFree(mA);
	hipFree(mB);
	hipFree(mC);
}

int compareMatrix (float *A, float *B,int n, int m)
{
	int size = n * m;
	for (int i = 0; i < size; i++ )
	{
	if (A[i] != B[i])
	{
		cout<<"the sequential result and parallel result are not equal"<<endl;
		return 0;
	}
	}
	cout<<"the sequential result and parallel result are equal"<<endl;
	return 0;
}

int main(int argc, char* argv[])
{
	clock_t start, finish;
	double elapsedsequential, elapsedParallel, elapsedParallelTiles, optimizationP, optimizationT;
	int kkkkk = atoi(argv[1]);
	int n = kkkkk;
	int m = kkkkk;
	int o = kkkkk;

	float *matA = (float *) malloc(n * m * sizeof(float));
	float *matB = (float *) malloc(m * o * sizeof(float));
	float *matCS = (float *) malloc(n * o * sizeof(float));
	float *matCP = (float *) malloc(n * o * sizeof(float));
	float *matCPT = (float *) malloc(n * o * sizeof(float));

	init_matrix(matA,1.5,n,m);
	init_matrix(matB,1.5,m,o);
	init_matrix(matCS,0,n,o);
	init_matrix(matCP,0,n,o);
	init_matrix(matCPT,0,n,o);


	start = clock();
	multMatrixSeq(matA,matB,matCS,n,m,o);
	finish = clock();
	elapsedsequential = (((double) (finish - start)) / CLOCKS_PER_SEC );
	cout<< "sequential matrix multiplication: " << elapsedsequential << "sec"<< endl<< endl;

	start = clock();
	multMatrixThread(matA,matB,matCP,n,m,o);
	finish = clock();
	elapsedParallel = (((double) (finish - start)) / CLOCKS_PER_SEC );
	cout<< "parallel matrix multiplication without using Tiles: " << elapsedParallel << "sec"<< endl<< endl;

	start = clock();
	multMatrixTiled(matA,matB,matCPT,n,m,o);
	finish = clock();
	elapsedParallelTiles = (((double) (finish - start)) / CLOCKS_PER_SEC );
	cout<< "parallel matrix multiplication using Tiles: " << elapsedParallelTiles << "sec"<< endl<< endl;

	optimizationP = elapsedsequential/elapsedParallel;
	cout<< "speedup without using Tiles: " << optimizationP <<endl;

	optimizationT = elapsedsequential/elapsedParallelTiles;
	cout<< "speedup using Tiles: " << optimizationT <<endl;

	cout<< "check parallel result without using Tiles: " <<endl;
	compareMatrix(matCS,matCP,n,o);
	cout<< "check parallel result using Tiles: " <<endl;
	compareMatrix(matCS,matCPT,n,o);
	
	if (DEBUG)
	{
		print(matCS,n,o);
		cout<<endl;
		print(matCP,n,o);
		cout<<endl;
		print(matCPT,n,o);
	}

	free (matA);
	free (matB);
	free (matCS);
	free (matCP);
	free (matCPT);
	return 0;
}
