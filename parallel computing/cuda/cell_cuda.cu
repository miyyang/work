
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>


static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

template< typename T >
void swap( T& a, T& b ) {
    T t = a;
    a = b;
    b = t;
}

struct DataBlock
{
int *outbitmap;
int *dev_in;
int *dev_out;
int *bitmap;
};

__global__ void update(int *in, int *out, int dim){
    int offset = threadIdx.x + blockIdx.x * blockDim.x;
    int x = offset % dim;
    int y = (int)(offset / dim);
    while (offset < dim * dim) {
        int sum = 0;
        for(int i=-1; i < 2; i++) {
            for(int j=-1; j < 2; j++) {
                int xtemp = (x + i + dim) % dim;
                int ytemp = (y + j + dim) % dim;
                int offsettemp = xtemp + ytemp * dim;
                sum = sum + in[offsettemp];
            }
        }
        sum = sum - in[offset];
        if (in[offset] == 1) {
            if (sum == 2 || sum ==3) {
                out[offset] = 1;
            }
            else {
                out[offset] = 0;
            }
        }
        else {
            if( sum == 3) {
                out[offset] = 1;
            }
            else {
                out[offset] = 0;
            }
        }
        offset = offset + blockDim.x * gridDim.x;
    }
}

int main(int argc, char *argv[]) {
    
    clock_t start;
    clock_t gpu_start;
    float gpu_comp_time = 0;
    float gpu_mem_to_time = 0, gpu_mem_back_time=0;
    int dim = atoi(argv[1]);
    int nStep = atoi(argv[2]);
//    int frequency = atoi(argv[3]);
    int size = dim * dim;
    int step;
    DataBlock data;
    data.bitmap=(int *)malloc(size * sizeof(int));
    for (int i = 0; i < size; i++) {
        data.bitmap[i] = 0;
    }
    data.bitmap[1]=1;
    data.bitmap[dim+2] = 1;
    data.bitmap[2 * dim + 0] = 1;
    data.bitmap[2 * dim + 1] = 1;
    data.bitmap[2 * dim + 2] = 1;
    data.outbitmap=(int *)malloc(size * sizeof(int));
    int bitmapSize=size * sizeof(int);
    
    start=clock();
    
    gpu_start = clock();
    HANDLE_ERROR(hipMalloc( (void **)&(data.dev_in), bitmapSize));
    HANDLE_ERROR(hipMalloc( (void **)&(data.dev_out), bitmapSize));
    HANDLE_ERROR(hipMemcpy(data.dev_in, data.bitmap, bitmapSize, hipMemcpyHostToDevice));
    gpu_mem_to_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    
    // dim3 dimgrid(dim / 16, dim / 16);
    // dim3 dimblock(16, 16);
    int grid_dim;
    int block_dim;
    if (dim < 1024) {
        grid_dim = dim;
        block_dim = dim;
    }
    else {
        grid_dim = 1024;
        block_dim = 1024;
    }
    
    gpu_start = clock();

    for(step = 0; step < nStep; step++ ){
        
        update<<<grid_dim, block_dim>>>(data.dev_in, data.dev_out,dim);
        
        swap(data.dev_in,data.dev_out);
    }

    hipDeviceSynchronize();
    gpu_comp_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    gpu_start = clock();
    HANDLE_ERROR(hipMemcpy(data.outbitmap, data.dev_out, bitmapSize, hipMemcpyDeviceToHost));
    gpu_mem_back_time = ((float)(clock() - gpu_start)) / CLOCKS_PER_SEC;
    HANDLE_ERROR(hipFree(data.dev_out));
    HANDLE_ERROR(hipFree(data.dev_in));
    
    printf("%f %f %f ", gpu_comp_time, gpu_mem_to_time, gpu_mem_back_time);
    printf("%f\n", ((float)(clock() - start)) / CLOCKS_PER_SEC);
}