#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

__global__ void generatePrimes(int *sizeD, int *maxPrimeD, int *numPrimesD) {
	__shared__ int numPrimesB;
	__shared__ int maxPrimeB;
	if(threadIdx.x == 0) {
		numPrimesB = 0;
		maxPrimeB = 0;
	}
	__syncthreads();
	int no = 9 + (blockIdx.x * 2000) + (2 * threadIdx.x);
	if(no < *sizeD) {
		int noRt = __double2int_ru(sqrt(__int2double_rn(no)));
		int k=3;
		for (k = 3; k <= noRt; k += 2) {
			if (no % k == 0) {
				break;
			}
		}
		if (k > noRt) {
			atomicAdd(&numPrimesB, 1);
			atomicMax(&maxPrimeB, no);
		}
		__syncthreads();
		if(threadIdx.x == 0) {
			atomicAdd(numPrimesD, numPrimesB);
			atomicMax(maxPrimeD, maxPrimeB);
		}
	}
}

int main(int argc, char* argv[]) {

	struct timeval t;
	double start_t, end_t, time_spent;
	int maxPrime = 0, numPrimes = 0, size = atoi(argv[1]);
	int gridSize = (int)(floor(size/2000.0));
	dim3 dimGrid(gridSize+1);
	dim3 dimBlock(1000);

	int *sizeD, *maxPrimeD, *numPrimesD;
	hipMalloc((void**)&sizeD, sizeof(int));
	hipMalloc((void**)&maxPrimeD, sizeof(int));
	hipMalloc((void**)&numPrimesD, sizeof(int));

	gettimeofday(&t, NULL);
	start_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	hipMemcpy(sizeD, &size, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(maxPrimeD, &maxPrime, sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(numPrimesD, &numPrimes, sizeof(int),hipMemcpyHostToDevice);
	generatePrimes<<<dimGrid, dimBlock>>> (sizeD, maxPrimeD, numPrimesD);
	hipMemcpy(&maxPrime, maxPrimeD, sizeof(int),hipMemcpyDeviceToHost);
	hipMemcpy(&numPrimes, numPrimesD, sizeof(int),hipMemcpyDeviceToHost);
	gettimeofday(&t, NULL);
	end_t = (t.tv_sec * 1000000.0) + t.tv_usec;
	time_spent = end_t - start_t;

	hipFree(sizeD);
	hipFree(maxPrimeD);
	hipFree(numPrimesD);

	
	printf("Max prime number %d\ninput size %d\nTotal time %f secs\nprime number %d\n",
			maxPrime, size, time_spent / 1000000.0, numPrimes+4);
	
	return 0;
}