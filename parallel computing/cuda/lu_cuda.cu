#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

__global__ void add( float *a, float *b, float *c) {
	int tid = blockIdx.x;
	c[tid] = a[tid] + b[tid];
}


__global__ void scale(float *a, int size, int index){
 	int i;
	int start=(index*size+index);
	int end=(index*size+size);
	
	for(i=start+1;i<end;i++)
		a[i]=(a[i]/a[start]);
}

__global__ void reduce(float *a, int size, int index, int b_size){
	extern __shared__ float pivot[];
	int i;

	int tid=threadIdx.x;
	int bid=blockIdx.x;
	int block_size=b_size;

	int start;
	int end;
	int pivot_row;
	int my_row;

	if(tid==0)
		for(i=index;i<size;i++) pivot[i]=a[(index*size)+i];

	__syncthreads();

	pivot_row=(index*size);
	my_row=(((block_size*bid) + tid)*size);
	start=my_row+index;
	end=my_row+size;

	if(my_row >pivot_row)
		for(i=start+1;i<end;i++)
			a[i]=a[i]-(a[start]*pivot[(i-my_row)]);
}

float a[5001][5001];
float b[5001][5001];
float c[5001][5001];
float result[5001][5001];


int main(int argc, char *argv[]){
	int N;
	int blocks;
	float *dev_a;
	int i;
	int j;
	N = atoi(argv[1]);	
	clock_t start, finish;
	double elapse;

	hipMalloc((void**)&dev_a, N * N * sizeof(float));
	srand((unsigned)2);
	for (i = 0; i <= N; i++)
		for (j = 0; j <= N; j++)
		a[i][j] = ((rand() % 10) + 1);
	hipMemcpy(dev_a, a, N * N * sizeof(float), hipMemcpyHostToDevice);

	start = clock();
	for(i = 0; i < N; i++){
		scale<<<1,1>>>(dev_a, N, i);
		blocks = N / 50;
		reduce<<<blocks,50,N*sizeof(float)>>>(dev_a, N, i, 50);
	}
 	finish = clock();
	elapse = (((double) (finish - start)) / CLOCKS_PER_SEC);
	printf("lu decompositon: %lf sec\n", elapse);

	hipMemcpy(c, dev_a, N * N * sizeof(float),hipMemcpyDeviceToHost);
	hipFree( dev_a );
	
	return 0;
}